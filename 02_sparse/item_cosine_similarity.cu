#include "hip/hip_runtime.h"
#include <stdbool.h>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <cmath>

/* CUDA libraries */
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "hugepages/thp.h"  // huge pages allocation
#include "definitions.h"


/***********************************************
 * Operations for loading matrices and vectors *
 **********************************************/

/* Load the ratings matrix as a sparse CSR matrix */
static void load_ratings_from_mtx(
    const char *fname,
    SparseMatrix dataset)
{
    int i=0, row=0, col=0;
    value_type rating=0.;
    FILE *fstream = fopen(fname, "r");

    if (fstream == NULL) {
        fprintf(stderr, "Error opening the file %s\n", fname);
        exit(EXIT_FAILURE);
    }

    if(fscanf(fstream, "%d %d %d", &dataset->nrows, 
                &dataset->ncols, &dataset->nnz) != 3) {
        fprintf(stderr, "The file is not valid\n");
        exit(EXIT_FAILURE);
    }

    dataset->data = (value_type *) alloc(dataset->nnz, sizeof(value_type));
    dataset->colInd = (int *) alloc(dataset->nnz, sizeof(int));
    dataset->rowPtr = (int *) alloc(dataset->nrows + 1, sizeof(int));
    assert(dataset->data && dataset->colInd && dataset->rowPtr);
    memset(dataset->rowPtr, 0, dataset->nrows + 1 * sizeof(int));

    for (i = 0; i < dataset->nnz; ++i) {
#ifdef DOUBLE
        if(fscanf(fstream, "%d %d %lf", &row, &col, &rating) != 3)
#else
        if(fscanf(fstream, "%d %d %f", &row, &col, &rating) != 3)
#endif
        {
            fprintf(stderr, "The file is not valid\n");
            exit(EXIT_FAILURE);
        }

        dataset->data[i] = rating;
        dataset->colInd[i] = col - 1;
        dataset->rowPtr[row] += 1;
    }
    
    for(row = 1; row < dataset->nrows + 1; ++row)
        dataset->rowPtr[row] += dataset->rowPtr[row-1];

    fclose(fstream);
} 


/* Load the correction vector from the given file */
static void load_correction_vector(
    const char *fname,
    value_type *correction_vector,
    const int vector_size)
{
    int i, read;
    FILE *fstream = fopen(fname, "r");
    
    if(fstream == NULL) {
        fprintf(stderr, "Error opening the file %s\n", fname);
        exit(EXIT_FAILURE);
    }   
    
    for(i = 0; i < vector_size; i++) {
#ifdef DOUBLE
        read = fscanf(fstream, "%le", &correction_vector[i]);
#else
        read = fscanf(fstream, "%e", &correction_vector[i]);
#endif
        if(read == EOF) {
            fprintf(stderr, "Error while reading file %s in element # %d\n", 
                    fname, i);
            fclose(fstream);
            exit(EXIT_FAILURE);
        }
    }

    fclose(fstream);
}


/************************************
 * Cosine similarity operations CPU *
 ***********************************/

/* Returns the vector representing the upper side of the similarity matrix 
 * by measuring cosine similarity pairwise for each row of the item/user matrix */
static void item_cosine_similarity(
    const int *item_user_matrix,
    value_type *similarity_matrix,
    const Dataset dataset)
{
    unsigned int i, u, v, uv, ui, vi;
    value_type num, uden, vden;

    for(u=0; u < dataset->items; u++) {
        for(v=u; v < dataset->items; v++) {
            num=0.;
            uden=0.;
            vden=0.;
            uv = (dataset->items * u) + v - u * (u+1) / 2; 

            for(i = 0; i < dataset->users; i++) {
                ui = u * dataset->users + i;
                vi = v * dataset->users + i;
                num += (value_type) (item_user_matrix[ui] * item_user_matrix[vi]);
                uden += (value_type) (item_user_matrix[ui] * item_user_matrix[ui]);
                vden += (value_type) (item_user_matrix[vi] * item_user_matrix[vi]);
            }
 
            similarity_matrix[uv] = num / (sqrt(uden) * sqrt(vden));
        }
    }
}


/************************************
 * Cosine similarity operations GPU *
 ***********************************/

/* CUDA version. Each thread is in charge of a pair of rows */
__global__ void item_cosine_similarity_cuda(
    const int *item_user_matrix,
    value_type *similarity_matrix,
    const int items,
    const int users)
{
    int i, ui, vi, uv;
    int u = blockIdx.y * blockDim.y + threadIdx.y;
    int v = blockIdx.x * blockDim.x + threadIdx.x;
    value_type num=0., uden=0., vden=0.;

    uv = items * u + v - u * (u + 1) / 2;

    if(v < u || u >= items || v >= items) return;

    for(i=0; i<users; i++) {
        ui = u * users + i;
        vi = v * users + i;
        num += (value_type) (item_user_matrix[ui] * item_user_matrix[vi]);
        uden += (value_type) (item_user_matrix[ui] * item_user_matrix[ui]);
        vden += (value_type) (item_user_matrix[vi] * item_user_matrix[vi]);
    }

    similarity_matrix[uv] = num * rsqrt(uden) * rsqrt(vden);
}


/*****************
 * Main function *
 ****************/

int main(int argc, char **argv) {
    bool correct=true;
    unsigned int i, num_iterations, distance_matrix_size;
    double startTime=0., 
           currentTime=0., 
           refTimeMean=0., 
           optTime=0., 
           previousMean=0.,
           cpuTime=0.,
           globalTime=0.,
           thisTime=0.;
    SparseMatrix dataset;
    value_type *correction_vector, *similarity_matrix, *d_similarity_matrix;

    if (argc < 3 || argc > 4) {
        fprintf(stderr, 
            "usage: ./item_cosine_similarity <user_item_rating_mtx>\
            <correction_vector_vec> [<no_of_iterations>]\n"
        );
        exit(EXIT_FAILURE);
    }
    
    /* start measuring time */
    //thisTime = omp_get_wtime();

    /* Useful for removing noise given by the usage of the machine */
    num_iterations = (argc == 4) ? atoi(argv[3]) : 1;

    /* Reserve space for the SparseMatrix basic structure */
    dataset = (SparseMatrix) malloc(sizeof(struct sDataset));
 
    /* Load ratings dataset from the given mtx file */
    debug("Loading ratings matrix from file %s\n", argv[1]);
    load_ratings_from_mtx(argv[1], dataset);
    debug("Successfully loaded %d total ratings of %d users and %d items\n", 
            dataset->nnz, dataset->ncols, dataset->nrows);

    for(int i=0; i<dataset->nnz; i++)
        fprintf(stderr, "%i ", dataset->data[i]);
    fprintf(stderr, "\n");
    for(int i=0; i<dataset->nnz; i++)
        fprintf(stderr, "%i ", dataset->colInd[i]);
    fprintf(stderr, "\n");
    for(int i=0; i<dataset->nrows + 1; i++)
        fprintf(stderr, "%i ", dataset->colInd[i]);
    fprintf(stderr, "\n");

    free(dataset->data);
    free(dataset->colInd);
    free(dataset->rowPtr);
    free(dataset);

//    /* We use a vector (representing the upper side of a triangular matrix) 
//       in order to make the correction */
//    distance_matrix_size = dataset->items * (dataset->items + 1) / 2;
//    correction_vector = (value_type *) alloc(distance_matrix_size, sizeof(value_type));
//    assert(correction_vector);
//    debug("Loding the correction vector from file %s\n", argv[2]);
//    load_correction_vector(argv[2], correction_vector, distance_matrix_size);
// 
//    /* Calculate the similarity matrix row-wise from the item/user matrix. 
//     * The matrix is represented by a vector of the upper triangular side.
//     * This is what I want to optimize */
//    similarity_matrix = (value_type *) alloc(distance_matrix_size, sizeof(value_type));
//    debug("Calculating items cosine similarity matrices of %d elements\n", 
//            dataset->items);
//
//  cpuTime = omp_get_wtime() - thisTime;
//    globalTime = cpuTime;
//
//    debug("Reference computation will run %d iterations\n", num_iterations);
//
//    for(i = 1; i <= num_iterations; i++) {
//        debug("\rReference iteration number # %d (%d left)", i, num_iterations-i);
//        
//        startTime = omp_get_wtime();
// 
//        /*  What I want to optimize */
//        item_cosine_similarity(item_user_matrix, similarity_matrix, dataset);
//        
//        currentTime = omp_get_wtime() - startTime;
//        previousMean = refTimeMean;
//        refTimeMean += 1.0/(double) i * (currentTime-previousMean);
//    }
//    debug("\nReference computation took %s%.5e%s s, plus %s%.5e%s for the setup.\n", 
//            YELLOW_TEXT, refTimeMean, NO_COLOR, YELLOW_TEXT, cpuTime,
//            NO_COLOR);
//
//    /* CUDA Setup */
//
//    thisTime = omp_get_wtime();
//
//    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
//    dim3 dimGrid((dataset->items + dimBlock.x - 1)/dimBlock.x,
//            (dataset->items + dimBlock.y - 1)/dimBlock.y, 1);
//    checkCudaErrors(hipMalloc(&d_item_user_matrix, 
//                dataset->items * dataset->users * sizeof(int)));
//    checkCudaErrors(hipMalloc(&d_similarity_matrix, 
//                distance_matrix_size * sizeof(value_type)));
//    assert(d_item_user_matrix && d_similarity_matrix);
//    checkCudaErrors(hipMemcpy(d_item_user_matrix, item_user_matrix, 
//                dataset->items * dataset->users * sizeof(int), 
//                hipMemcpyDefault));
//
//    globalTime += omp_get_wtime() - thisTime;
//
//    debug("Running optimized computation\n");
// 
//    /* Optimized computation */
//        
//    startTime = omp_get_wtime();
//
//    /* Run cuda kernel */
//    checkCudaErrors(hipMemset(d_similarity_matrix, 0.0f, 
//                distance_matrix_size * sizeof(value_type)));
//    item_cosine_similarity_cuda<<< dimGrid, dimBlock >>>(d_item_user_matrix, 
//            d_similarity_matrix, (int) dataset->items, (int) dataset->users);
//    getLastCudaError("item_cosine_similarity_cuda() kernel failed");
//    checkCudaErrors(hipDeviceSynchronize());
// 
//    optTime = omp_get_wtime() - startTime;
//
//    thisTime = omp_get_wtime();
//    checkCudaErrors(hipMemcpy(similarity_matrix, d_similarity_matrix, 
//                distance_matrix_size * sizeof(value_type), hipMemcpyDefault));
//    globalTime = omp_get_wtime() - thisTime;
//    
//    debug("Optimized computation took %s%.5e%s s plus %s%.5e%s "
//            "for the setup.\n", 
//            YELLOW_TEXT, optTime, NO_COLOR, YELLOW_TEXT, globalTime,
//            NO_COLOR);
//    debug("Rough calculations time speedup: %s%.2fx%s\n",
//          BLUE_TEXT, (refTimeMean)/(optTime), NO_COLOR);
//    debug("Rough wall time speedup: %s%.2fx%s\n",
//          BLUE_TEXT, (refTimeMean+cpuTime)/(optTime+globalTime), NO_COLOR);
// 
//    /* Correction using the previously loaded correction vector */
//    debug("Correction using the given vector and an error of %.0e\n", ERROR);
//    for(i = 0; i < distance_matrix_size; i++) {
//        if(fabs(similarity_matrix[i] - correction_vector[i]) >= ERROR) {
//            correct = false;
//#ifdef DEBUG
//            fprintf(stdout, "%d %.5e %.5e %.5e\n", i, similarity_matrix[i], 
//                    correction_vector[i], 
//                    fabs(similarity_matrix[i] - correction_vector[i]));
//            fflush(stdout);
//#endif
//        }
//    }
//    if(correct){
//        debug("Calculations were %s%s%s\n", GREEN_TEXT, "CORRECT", NO_COLOR);
//    } else {
//        debug("Calculations were %s%s%s\n", RED_TEXT, "WRONG", NO_COLOR);
//    }
//
//    free(dataset);
//    free(ratings);
//    free(item_user_matrix);
//    free(similarity_matrix);
//    free(correction_vector);
//  checkCudaErrors(hipFree(d_item_user_matrix));
//  checkCudaErrors(hipFree(d_similarity_matrix));
//
    return EXIT_SUCCESS;
}
